#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <chrono>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

inline int findCudaDevice(int argc, const char **argv, bool print)
{
  hipDeviceProp_t deviceProp;
  int devID = 0;

  // If the command-line has a device number specified, use it
  if (checkCmdLineFlag(argc, argv, "device"))
  {
    devID = getCmdLineArgumentInt(argc, argv, "device=");
    if (devID < 0){
      printf("Invalid command line parameter\n ");
      exit(EXIT_FAILURE);
    }
    else{
      devID = gpuDeviceInit(devID);
      if (devID < 0){
        printf("exiting...\n");
        exit(EXIT_FAILURE);
      }
    }
  }
  else
  {
    // Otherwise pick the device with highest Gflops/s
    devID = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    if (print) printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  return devID;
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <typename T, int BLOCK_SIZE> __global__ void MatrixMulCUDA(T *C, T *A, T *B, int wA, int wB) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  T Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ T As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ T Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
    #pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

template <typename T> void ConstantInit(T *data, int size, T val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
template <typename T> int MatrixMultiply(int argc, char **argv, int block_size, const dim3 &dimsA, const dim3 &dimsB, bool print) {
  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(T) * size_A;
  T *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(T) * size_B;
  T *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  hipStream_t stream;

  // Initialize host memory
  const T valA = 1.00f;
  const T valB = 0.01f;
  ConstantInit<T>(h_A, size_A, valA);
  ConstantInit<T>(h_B, size_B, valB);

  // Allocate device memory
  T *d_A, *d_B, *d_C;

  // Allocate host matrix C
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(T);
  T *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

  if (h_C == NULL) {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // copy host memory to device
  checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

  // Setup execution parameters
  dim3 threads(block_size, block_size);
  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

  // Create and start timer
  if (print) printf("Computing result using CUDA Kernel...\n");

  // Performs warmup operation using matrixMul CUDA kernel
  switch(block_size){
    case 1:
      MatrixMulCUDA<T, 1><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 2:
      MatrixMulCUDA<T, 2><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 4:
      MatrixMulCUDA<T, 4><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 8:
      MatrixMulCUDA<T, 8><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 16:
      MatrixMulCUDA<T, 16><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    default:
      MatrixMulCUDA<T, 32><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }

  if (print) printf("done\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  // Record the start event
  auto chrono_start = std::chrono::steady_clock::now();

  // Execute the kernel
  int nIter = 300;

  switch(block_size){
    case 1:
      for (int j = 0; j < nIter; j++)
        MatrixMulCUDA<T, 1><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 2:
      for (int j = 0; j < nIter; j++)
        MatrixMulCUDA<T, 2><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 4:
      for (int j = 0; j < nIter; j++)
        MatrixMulCUDA<T, 4><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 8:
      for (int j = 0; j < nIter; j++)
        MatrixMulCUDA<T, 8><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case 16:
      for (int j = 0; j < nIter; j++)
        MatrixMulCUDA<T, 16><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    default:
      for (int j = 0; j < nIter; j++)
        MatrixMulCUDA<T, 32><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }

  // Wait for the stop event to complete
  checkCudaErrors(hipStreamSynchronize(stream));

  // Record the stop event
  auto chrono_stop = std::chrono::steady_clock::now();

  double chrono_msecTotal = std::chrono::duration_cast<std::chrono::milliseconds>(chrono_stop - chrono_start).count();

  // Compute and print the performance
  double msecPerMatrixMul = chrono_msecTotal / nIter;
  double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) * static_cast<double>(dimsA.y) * static_cast<double>(dimsB.x);
  double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  if (print) printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
    " WorkgroupSize= %u threads/block\n",
    gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);
  if (print) printf("Block_size=%d\n", block_size);

  if(!print) printf("%d, %d, %d, %d, %d, %.2f, %.3f, %.1f, %u\n", 
    block_size, dimsA.x, dimsA.y, dimsB.x, dimsB.y, gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);

  // Copy result from device to host
  checkCudaErrors(hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  if (print) printf("Checking computed result for correctness: ");
  bool correct = true;

  // test relative error by the formula
  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1.e-6;  // machine zero

  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;

    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x * valB, eps);
      correct = false;
    }
  }

  if (print) printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

  // Clean up memory
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  if (print) printf("NOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

  if (correct) {
    return EXIT_SUCCESS;
  } else {
    return EXIT_FAILURE;
  }
}


/**
 * Program main
 */
int main(int argc, char **argv) {
  if (checkCmdLineFlag(argc, (const char **)argv, "help") || checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -bs=m (Size of the block = 1 << (5 - m))\n");
    printf("      -dt=datatype (Type of the data 0=int, 1=float, 2=double)\n");
    printf("      -sM=s (Size of the square matrixes)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("Note: Outer matrix dimensions of A & B matrices must be equal.\n");
    exit(EXIT_SUCCESS);
  }

  bool print = false;

  // extended results printing
  if (checkCmdLineFlag(argc, (const char **)argv, "p")) {
    print = true;
  }

  if (print) printf("[Matrix Multiply Using CUDA] - Starting...\n");

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line
  int dev = findCudaDevice(argc, (const char **)argv, print);

  int block_size = 32;

  // size of the block
  if (checkCmdLineFlag(argc, (const char **)argv, "bs")) {
    block_size = 1 << (5 - getCmdLineArgumentInt(argc, (const char **)argv, "bs"));
  }

  dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
  dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

  // size of squared matrixes
  if (checkCmdLineFlag(argc, (const char **)argv, "sM")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "sM");
    dimsA.y = dimsA.x;
    dimsB.x = dimsA.x;
    dimsB.y = dimsA.x;
  }

  // width of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  // height of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  // width of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  // height of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  if (dimsA.y % block_size != 0 || dimsB.x % block_size != 0) {
    printf("Error: internal matrix dimensions must be divisible by block_size.\n");
    exit(EXIT_FAILURE);
  }

  if (print) printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  checkCudaErrors(hipProfilerStart());

  int matrix_result = EXIT_FAILURE;

  // type of the data
  if (checkCmdLineFlag(argc, (const char **)argv, "dt")) {
    switch(getCmdLineArgumentInt(argc, (const char **)argv, "dt")){
      case 0:
        if (!print) printf("int, ");
        matrix_result = MatrixMultiply<int>(argc, argv, block_size, dimsA, dimsB, print);
        break;
      case 2:
        if (!print) printf("double, ");
        matrix_result = MatrixMultiply<double>(argc, argv, block_size, dimsA, dimsB, print);
        break;
      default:
        if (!print) printf("float, ");
        matrix_result = MatrixMultiply<float>(argc, argv, block_size, dimsA, dimsB, print);
    }
  }
  checkCudaErrors(hipProfilerStop());

  exit(matrix_result);
}
